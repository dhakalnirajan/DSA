#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

__device__ float quake3InvSqrt(float x) {
    float xhalf = 0.5f * x;
    int i = *(int*)&x; // store floating-point bits in integer
    i = 0x5f3759df - (i >> 1); // initial guess
    float y = *(float*)&i;
    return y;
}

__global__ void isrKernel(float *x, float *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = quake3InvSqrt(x[idx]);
}

int main() {
    float x = 2.0f;
    float *d_x, *d_result;
    hipMalloc((void**)&d_x, sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));

    hipMemcpy(d_x, &x, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = 1;
    isrKernel<<<numBlocks, blockSize>>>(d_x, d_result);

    float result;
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_result);

    printf("Quake III Inverse Square Root of %f: %f\n", x, result);

    return 0;
}