#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

__global__ void simdInvSqrtKernel(float *x, float *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __half hx = __half(x[idx]);
    __half hy = __hrsqrt(hx);
    hy = __hmul(hy, __half(1.5f));
    hy = __hsub(hy, __hmul(hx, __hmul(hy, hy)));
    result[idx] = __half2float(hy);
}

int main() {
    float x = 2.0f;
    float *d_x, *d_result;
    hipMalloc((void**)&d_x, sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));

    hipMemcpy(d_x, &x, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = 1;
    simdInvSqrtKernel<<<numBlocks, blockSize>>>(d_x, d_result);

    float result;
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_result);

    printf("SIMD-optimized Inverse Square Root of %f: %f\n", x, result);

    return 0;
}