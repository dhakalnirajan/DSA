#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

__device__ float newtonRaphsonInvSqrt(float x, int iterations) {
    float y = x;
    for (int i = 0; i < iterations; i++) {
        y = y * (1.5f - (x * 0.5f * y * y));
    }
    return y;
}

__global__ void isrKernel(float *x, float *result, int iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = newtonRaphsonInvSqrt(x[idx], iterations);
}

int main() {
    float x = 2.0f;
    int iterations = 5;
    float *d_x, *d_result;
    hipMalloc((void**)&d_x, sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));

    hipMemcpy(d_x, &x, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = 1;
    isrKernel<<<numBlocks, blockSize>>>(d_x, d_result, iterations);

    float result;
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_result);

    printf("Newton-Raphson Inverse Square Root of %f: %f\n", x, result);

    return 0;
}