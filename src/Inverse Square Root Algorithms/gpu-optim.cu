#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

__global__ void gpuIsrKernel(float *x, float *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = rsqrtf(x[idx]);
}

int main() {
    float x = 2.0f;
    float *d_x, *d_result;
    hipMalloc((void**)&d_x, sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));

    hipMemcpy(d_x, &x, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = 1;
    gpuIsrKernel<<<numBlocks, blockSize>>>(d_x, d_result);

    float result;
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_result);

    printf("GPU-optimized Inverse Square Root of %f: %f\n", x, result);

    return 0;
}